#include "hip/hip_runtime.h"
#include <OPP/OPP_cuda_buffer.cuh>
#include <OPP/OPP_cuda_histogram.cuh>
#include <exo6/student.h>

#include <OPP_cuda.cuh>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <type_traits>

#include <hip/hip_runtime.h>
#include <cmath>
#include <OPP/OPP_cuda_buffer.cuh>
#include <OPP/OPP_cuda_histogram.cuh>
#include <exo6/student.h>

#include <OPP_cuda.cuh>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <iostream>
#include <type_traits>

namespace {

  /**
   * @brief print a device buffer on standard output
   *
   * @tparam T Datatype of the data stored into the DeviceDuffer
   * @param msg Message to display first (before the data)
   * @param deviceBuffer Contains the data to display
   */
  template <typename T>
  void print(std::string &msg, OPP::CUDA::DeviceBuffer<T> &deviceBuffer)
  {
    const auto size = deviceBuffer.getNbElements();
    std::vector<T> hostVector(size);
    deviceBuffer.copyToHost(hostVector.data());
    std::cout << "======= " << msg << " of size " << size << " =====\n";
    for (unsigned i{0u}; i < size; ++i) {
      std::cout << hostVector[i] << " ";
      if ((i % 16u) == 15u) {
        std::cout << "\n";
      }
    }
  }

  using uchar = unsigned char;

  __global__
      void buildHistogramAndVarianceSum_kernel(
          const float* const dev_inputValue,
          unsigned* const dev_histo,
          float* const dev_weight,
          const unsigned size,
          const unsigned imageWidth) {

    __shared__ float shared_sum;
    __shared__ unsigned shared_count;
    __shared__ unsigned shared_histo[257];

    shared_sum = 0.0f;
    shared_count = 0;

    for (int i = 0; i < 257; i++) {
      shared_histo[i] = 0;
    }

    const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned stride = blockDim.x * gridDim.x;

    for (unsigned i = tid; i < size; i += stride) {
      float pixel = dev_inputValue[i];
      shared_sum += pixel;
      shared_count++;

      const unsigned xi = min(static_cast<unsigned>(pixel), 256u);
      atomicAdd(&shared_histo[xi], 1u);

      const unsigned row = i / imageWidth;
      const unsigned col = i % imageWidth;
      const unsigned left_idx = row * imageWidth + ((col + imageWidth - 1) % imageWidth);
      const unsigned right_idx = row * imageWidth + ((col + 1) % imageWidth);

      const float left_value = dev_inputValue[left_idx];
      const float right_value = dev_inputValue[right_idx];
      const float local_variance = (left_value - pixel) * (left_value - pixel) +
                                   (right_value - pixel) * (right_value - pixel);
      atomicAdd(&dev_weight[xi], local_variance / 2.0f);
    }

    __syncthreads();

    if (threadIdx.x == 0) {
      dev_weight[256] = shared_sum;
      dev_weight[257] = shared_count;
    }

    __syncthreads();

    if (dev_weight[257] > 0) {
      float mean = dev_weight[256] / dev_weight[257];
      float sum_squared_diff = 0.0f;

      for (unsigned i = tid; i < size; i += stride) {
        float diff = dev_inputValue[i] - mean;
        sum_squared_diff += diff * diff;
      }

      atomicAdd(&dev_weight[258], sum_squared_diff * 256);
    }

    __syncthreads();

    for (unsigned i = threadIdx.x; i < 257; i += blockDim.x) {
      dev_histo[i] = shared_histo[i];
    }
  }

  __global__
      void buildCumulativeDistributionFunction_kernel(
          unsigned* const dev_cdf,
          const float* const dev_weight,
          const float lambda,
          const unsigned size) {

    const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < 256) {
      float sum_weights = 0.0f;
      for (unsigned i = 0; i <= tid; ++i) {
        sum_weights += dev_weight[i];
      }
      float normalized_weight = (sum_weights + lambda) / (size + 256.0f * lambda);
      dev_cdf[tid] = static_cast<unsigned>(normalized_weight * size);
    }
  }

  __global__
      void applyTransformation_kernel(
          const float* const dev_inputValue,
          const unsigned* const dev_cdf,
          float* const dev_outputValue,
          const unsigned size) {
    const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
      const uchar xi = uchar(dev_inputValue[tid]);
      const float cdf_sum = float(dev_cdf[255]);
      dev_outputValue[tid] = 255.f * float(dev_cdf[xi]) / cdf_sum;
    }
  }


  void buildHistogramAndVarianceSum(OPP::CUDA::DeviceBuffer<float> &dev_inputValue,
                                    OPP::CUDA::DeviceBuffer<unsigned> &dev_cdf,
                                    OPP::CUDA::DeviceBuffer<float> &dev_weight,
                                    const unsigned imageWidth)
  {
    const unsigned nbThreads = 1024;
    const unsigned size = dev_inputValue.getNbElements();

    const dim3 threads(nbThreads);
    const dim3 blocks((size + nbThreads - 1) / nbThreads);

    buildHistogramAndVarianceSum_kernel<<<blocks, threads>>>(
        dev_inputValue.getDevicePointer(),
        dev_cdf.getDevicePointer(),
        dev_weight.getDevicePointer(),
        size,
        imageWidth);
    hipDeviceSynchronize();
  }

  void buildCumulativeDistributionFunction(OPP::CUDA::DeviceBuffer<unsigned> &dev_cdf,
                                           OPP::CUDA::DeviceBuffer<float> &dev_weight,
                                           const float lambda,
                                           const unsigned size)
  {
    const unsigned nbThreads = 1024;

    const dim3 threads(nbThreads);
    const dim3 blocks((size + nbThreads - 1) / nbThreads);

    buildCumulativeDistributionFunction_kernel<<<blocks, threads>>>(
        dev_cdf.getDevicePointer(),
        dev_weight.getDevicePointer(),
        lambda,
        size);
    hipDeviceSynchronize();
  }

  void applyTransformation(OPP::CUDA::DeviceBuffer<float> &dev_inputValue,
                           OPP::CUDA::DeviceBuffer<unsigned> &dev_cdf,
                           OPP::CUDA::DeviceBuffer<float> &dev_outputValue)
  {
    const unsigned nbThreads = 1024;
    const unsigned size = dev_inputValue.getNbElements();

    const dim3 threads(nbThreads);
    const dim3 blocks((size + nbThreads - 1) / nbThreads);

    applyTransformation_kernel<<<blocks, threads>>>(
        dev_inputValue.getDevicePointer(),
        dev_cdf.getDevicePointer(),
        dev_outputValue.getDevicePointer(),
        size);
    hipDeviceSynchronize();
  }
} // namespace

void StudentWorkImpl::run_WHE([[maybe_unused]] OPP::CUDA::DeviceBuffer<float> &dev_inputValue,
                              [[maybe_unused]] OPP::CUDA::DeviceBuffer<unsigned> &dev_histo,
                              [[maybe_unused]] OPP::CUDA::DeviceBuffer<float> &dev_weight,
                              [[maybe_unused]] OPP::CUDA::DeviceBuffer<float> &dev_outputValue,
                              const unsigned imageWidth,
                              const unsigned imageHeight,
                              const float lambda)
{
  // 1. calcul par valeur dans [0..255/256] de l'histogramme ET de la somme des variances/valeur
  ::buildHistogramAndVarianceSum(dev_inputValue, dev_histo, dev_weight, imageWidth);

  // ::print(std::string("histo"), dev_histo); // for debug, if needed
  // ::print(std::string("weight"), dev_weight); // for debug, if needed

  // 2. calcul de la CDF (dans histo pour économiser de la mémoire)
  ::buildCumulativeDistributionFunction(dev_histo, dev_weight, lambda, imageWidth * imageHeight);

  // 3. application de la transformation...
  ::applyTransformation(dev_inputValue, dev_histo, dev_outputValue);
}
/**********************************/
/*   AL NATOUR MAZEN, M1 Info CL  */
/**********************************/